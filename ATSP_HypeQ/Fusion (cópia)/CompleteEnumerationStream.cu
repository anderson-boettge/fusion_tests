/*
 * main.c
 *
 *  Created on: 26/01/2011
 *      Author: einstein/carneiro
 */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define mat(i,j) mat_h[i*N+j]
#define mat_h(i,j) mat_h[i*N+j]
#define mat_d(i,j) mat_d[i*N_l+j]
#define mat_block(i,j) mat_block[i*N_l+j]
#define proximo(x) x+1
#define anterior(x) x-1
#define MAX 8192
#define INFINITO 999999
#define ZERO 0
#define ONE 1

#define _VAZIO_      -1
#define _VISITADO_    1
#define _NAO_VISITADO_ 0

int qtd = 0;
int custo = 0;
int N;
int melhor = INFINITO;
int upper_bound;



#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }





#define HANDLE_NULL( a ) {if (a == NULL) { \
		printf( "Host memory failed in %s at line %d\n", \
				__FILE__, __LINE__ ); \
				exit( EXIT_FAILURE );}}

#ifndef _DFS_CUDA_UB_STREAM_H_
#define _DFS_CUDA_UB_STREAM_H_

#include <stdio.h>

#ifdef __cplusplus
extern "C"
{
#endif
__global__ void dfs_cuda_UB_stream(int N,int stream_size, int *mat_d, 
	short *preFixos_d, int nivelPrefixo, int upper_bound, int *sols_d,
	int *melhorSol_d)
	{

	register int idx = blockIdx.x * blockDim.x + threadIdx.x;
	register int flag[16];
	register int vertice[16]; 

	register int N_l = N;

	register int i, nivel;
	register int custo;
	register int qtd_solucoes_thread = 0;
	register int UB_local = upper_bound;
	register int nivelGlobal = nivelPrefixo;
	int stream_size_l = stream_size;

	if (idx < stream_size_l) {

		for (i = 0; i < N_l; ++i) {
			vertice[i] = _VAZIO_;
			flag[i] = _NAO_VISITADO_;
		}

		vertice[0] = 0;
		flag[0] = _VISITADO_;
		custo= ZERO;

		for (i = 1; i < nivelGlobal; ++i) {

			vertice[i] = preFixos_d[idx * nivelGlobal + i];

			flag[vertice[i]] = _VISITADO_;
			custo += mat_d(vertice[i-1],vertice[i]);
		}

		nivel=nivelGlobal;

		while (nivel >= nivelGlobal ) {
			if (vertice[nivel] != _VAZIO_) {
				flag[vertice[nivel]] = _NAO_VISITADO_;
				custo -= mat_d(vertice[anterior(nivel)],vertice[nivel]);
			}

			do {
				vertice[nivel]++;
			} while (vertice[nivel] < N_l && flag[vertice[nivel]]); 
			
			if (vertice[nivel] < N_l) {
				custo += mat_d(vertice[anterior(nivel)],vertice[nivel]);
				flag[vertice[nivel]] = _VISITADO_;
				nivel++;

				if (nivel == N_l) {
					++qtd_solucoes_thread;
					if (custo + mat_d(vertice[anterior(nivel)],0) < UB_local) {
						UB_local = custo + mat_d(vertice[anterior(nivel)],0);
					}
					nivel--;
				}
			}
			else {
				vertice[nivel] = _VAZIO_;
				nivel--;
			}
		}
		sols_d[idx] = qtd_solucoes_thread;
		melhorSol_d[idx] = UB_local;
	}
}
#ifdef __cplusplus
}
#endif
#endif

#ifdef __cplusplus
extern "C" {
#endif
  
int *mat_d, *mat_h;
int *qtd_threads_streams;

int block_size =192, nivelPreFixos;
int *sols_h, *sols_d; 
int *melhorSol_h, *melhorSol_d; 
short * path_h, * path_d;
int chunk;
int numStreams, nPreFixos;
hipStream_t vectorOfStreams[4];
int qtd_sols_global=0, otimo_global=INFINITO;

static void HandleError( hipError_t err,const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),
				file, line );
		exit( EXIT_FAILURE );
	}
}


void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int completeEnum(int* mat, int nivelPreF, int nPre, int tam, short* path, int nStream){
  mat_h = mat;
  //int nPreFixos = calculaNPrefixos(nivelPreFixos,N);
  nivelPreFixos = nivelPreF;
  
  nPreFixos = nPre;
  N = tam;
//   printf("nivelPreFixos: %d\nnPreFIxos: %d\nN: %d",nivelPreFixos, nPreFixos,N);
  
  chunk = nPreFixos/nStream; 
  numStreams = nStream;
//   printf("chunk: %d\nnStreams: %d\n\n\n", chunk,numStreams);
  
  qtd_threads_streams = (int*)malloc(sizeof(int)*numStreams);
  
  if(numStreams>1){
    for(int i = 0; i<numStreams-1 / block_size;++i){
      qtd_threads_streams[i] = chunk;
    }
  }
  
  CUDA_CHECK_RETURN( hipMalloc((void **) &path_d, nPreFixos*nivelPreFixos*sizeof(short)));
  
  sols_h = (int*)malloc(sizeof(int)*nPreFixos);
  melhorSol_h = (int*)malloc(sizeof(int)*nPreFixos); 
  
  CUDA_CHECK_RETURN( hipMalloc((void **) &mat_d, N * N * sizeof(int)));
  
  path_h = path;
  
  CUDA_CHECK_RETURN( hipMemcpy(mat_d, mat_h, N * N * sizeof(int), hipMemcpyHostToDevice));
  
  //for(int i =0; i<N*N;i++) printf("[ %d ]",mat_h[i]);
  
  for(int i = 0; i<nPreFixos; ++i) melhorSol_h[i] = INFINITO;
  
  CUDA_CHECK_RETURN( hipMalloc((void **) &melhorSol_d, sizeof(int)*nPreFixos));
  CUDA_CHECK_RETURN( hipMalloc((void **) &sols_d, sizeof(int)*nPreFixos));
  
    
}

int createStream(int rank){
    //printf("createStream: %d",rank);
    hipStreamCreate(&vectorOfStreams[rank]);
    
    hipMemcpyAsync(&path_d[rank*chunk*nivelPreFixos],&path_h[rank*chunk*nivelPreFixos],qtd_threads_streams[rank]*sizeof(short)*nivelPreFixos,hipMemcpyHostToDevice,vectorOfStreams[rank]);
    hipMemcpyAsync(&melhorSol_d[rank*chunk], &melhorSol_h[rank*chunk],qtd_threads_streams[rank]*sizeof(int), hipMemcpyHostToDevice, vectorOfStreams[rank]);
    hipMemcpyAsync(&sols_d[rank*chunk], &sols_h[rank*chunk], qtd_threads_streams[rank]*sizeof(int),hipMemcpyHostToDevice,vectorOfStreams[rank]);
    
    return rank;
}
int callCompleteEnumStreams(int rank){
	
	//int resto = 0;

	//resto = (nPreFixos % chunk);
	
	const int num_blocks = chunk/block_size + (chunk % block_size == 0 ? 0 : 1); //13: 16 blocos 
// 	printf("Kernel %d\nnumblocks: %d \nblocksize: %d\n",rank, num_blocks, block_size);
	
	dfs_cuda_UB_stream<<<num_blocks,block_size,0,vectorOfStreams[rank]>>>(N,qtd_threads_streams[rank],mat_d, &path_d[rank*chunk*nivelPreFixos],nivelPreFixos,999999, &sols_d[rank*chunk],&melhorSol_d[rank*chunk]);
	hipMemcpyAsync(&sols_h[rank*chunk],&sols_d[rank*chunk], qtd_threads_streams[rank]*sizeof(int),hipMemcpyDeviceToHost,vectorOfStreams[rank]);
	hipMemcpyAsync(&melhorSol_h[rank*chunk],&melhorSol_d[rank*chunk], qtd_threads_streams[rank]*sizeof(int),hipMemcpyDeviceToHost,vectorOfStreams[rank]);
	
	hipDeviceSynchronize();
	
// 	cudaMemcpy(sols_h,sols_d, nPreFixos*sizeof(int),cudaMemcpyDeviceToHost);
// 	cudaMemcpy(melhorSol_h,melhorSol_d, nPreFixos*sizeof(int),cudaMemcpyDeviceToHost);
// 	
// 	
	//testandoooooooooo
	if(rank==0){
	  for(int i = 0; i<nPreFixos; ++i){
	    qtd_sols_global+=sols_h[i];
	    if(melhorSol_h[i]<otimo_global)
	      otimo_global = melhorSol_h[i];
	  }

// 	  printf("\n\n\n\t niveis preenchidos: %d.\n",nivelPreFixos);
// 
// 	  printf("\t Numero de streams: %d.\n",numStreams);
// 	  printf("\t Tamanho do stream: %d.\n",chunk);
// 	  printf("\nQuantidade de solucoes encontradas: %d.", qtd_sols_global);
// 	  printf("\n\tOtimo global: %d.\n\n", otimo_global);
	}
	return rank;
}

int getQT(){
  return qtd_sols_global;
}


int getSol(){
  return otimo_global;
}

int clearAll(){
  CUDA_CHECK_RETURN( hipFree(mat_d));
  CUDA_CHECK_RETURN( hipFree(sols_d));
  CUDA_CHECK_RETURN( hipFree(path_d));
  CUDA_CHECK_RETURN( hipFree(melhorSol_d));
  
  for(int i = 0; i < numStreams; i++)  hipStreamDestroy(vectorOfStreams[i]);
  
  return 1;
}
#ifdef __cplusplus
}
#endif
