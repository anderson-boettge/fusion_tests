
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

__global__ void mult_mat(float *matA, float *matB, float *matR, int ncol, int nRows, int stream) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	float res=0;
	int nPr = stream*ncol*nRows;
	for(int i=0; i<ncol; i++){
	  res += matA[ncol*idy+i] * matB[i*ncol+idx];
	}
	matR[nPr+idy*ncol+idx] = res;
	if(stream==0)
	printf("[%d] %.2f ", nPr+idy*ncol+idx, matR[nPr+idy*ncol+idx]);
}

float *matA, *matR, *matB;
int numColA, numRowA, numColB, numRowB, nCore, nRows;
FILE *arq;

float* read(){
	int numCol, numRow;
	fscanf(arq,"%d",&numRow);
	fscanf(arq,"%d",&numCol);
	numColB = numCol;
	float *mat = (float*)malloc(sizeof(float)*numCol*numRow);
	for(int i=0;i<numCol*numRow;i++){
		fscanf(arq,"%f",&mat[i]);
	}
	fclose(arq);
	return mat;
}

void print(int rows){
	for(int i=0; i<numColB*rows; i++){
		if(i%(numColB) == 0)
			printf("\n");
		printf("%.2f ",matR[i]);
	}
}

void multiply(){
	/*Variaveis globais no dispositivo*/
	float *matA_d;
	hipMalloc((void **) &matA_d, numColA*numRowA*sizeof(float));
	float *matR_d;
	hipMalloc((void **) &matR_d, numColB*numRowA*sizeof(float));
	float *matB_d;
	hipMalloc((void **) &matB_d, numColB*numRowA*sizeof(float));

	hipStream_t vectorOfStreams[nCore];
	for(int stream_id=0; stream_id<nCore; stream_id++)
				hipStreamCreate(&vectorOfStreams[stream_id]);

	/*transferencia das matrizes A e B para GPU*/
	for(int stream_id=0; stream_id<nCore; stream_id++)
			hipMemcpyAsync(&matA_d[nRows*stream_id*numColA],&matA[nRows*stream_id*numColA],nRows*numColA*sizeof(float),
					hipMemcpyHostToDevice,vectorOfStreams[stream_id]);

	hipMemcpy(matB_d, matB, numColB*numRowB*sizeof(float),hipMemcpyHostToDevice);

	dim3 block(32,32);
    dim3 grid(numColB/32,(numRowA/4)/32);

    for(int stream_id=0; stream_id<nCore; stream_id++){
    	mult_mat<<<grid,block,0,vectorOfStreams[stream_id]>>>
    			(&matA_d[nRows*stream_id],matB_d,matR_d,numColB,nRows,stream_id);
    }
    hipDeviceSynchronize();
	/*copia da resultante para CPU*/
    for(int stream_id=0; stream_id<nCore; stream_id++){
    	hipMemcpyAsync(&matR[nRows*stream_id*numColB], &matR_d[nRows*stream_id*numColB],
    			numColB*nRows*sizeof(float),hipMemcpyDeviceToHost, vectorOfStreams[stream_id]);
    }
	hipDeviceSynchronize();

	hipFree(matA_d);
	hipFree(matB_d);
	hipFree(matR_d);
}

int main(int argc, char *argv[]) {
	printf("Aplicação -/- -/- -/- %s / %s",argv[1],argv[2]);
	arq = fopen(argv[1],"r");
	if (arq == NULL) {
		printf ("Houve um erro ao abrir o arquivo.\n");
		return 1;
	}
	matA = read();
	arq = fopen(argv[2],"r");
	if (arq == NULL) {
		printf ("Houve um erro ao abrir o arquivo.\n");
		return 1;
	}
	matB = read();
	numColA = numRowA = numRowB = numColB;
	arq = fopen("time.txt","a");
	struct timeval utime;
	double tstart, tend;

	gettimeofday(&utime, NULL);
	tstart = utime.tv_sec + ( utime.tv_usec / 1000000.0 );

	nCore = 4;
	nRows = numRowA/nCore;

	matR = (float*) malloc(sizeof(float)*numRowA*numColB);
	multiply();
	print(128);
	free(matA);
    free(matB);
    free(matR);

    gettimeofday(&utime, NULL);
    tend = utime.tv_sec + ( utime.tv_usec / 1000000.0 );

    printf("\n\nExecution time: %.4lf\n",tend-tstart);
    fprintf(arq, "%.4lf\n",tend-tstart);
    fclose(arq);
    return 0;
}
