
/*
 * main.c
 *
 *  Created on: 26/01/2011
 *      Author: einstein/carneiro
 */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define mat_h(i,j) mat_h[i*N+j]
#define mat_d(i,j) mat_d[i*N_l+j]
#define mat_block(i,j) mat_block[i*N_l+j]
#define proximo(x) x+1
#define anterior(x) x-1
#define MAX 8192
#define INFINITO 999999
#define ZERO 0
#define ONE 1

#define _VAZIO_      -1
#define _VISITADO_    1
#define _NAO_VISITADO_ 0

int qtd = 0;
int custo = 0;
int N;
int melhor = INFINITO;
int upper_bound;

int mat_h[MAX];


void read() {
	int i;
	//scanf("%d", &upper_bound);
	scanf("%d", &N);
	for (i = 0; i < (N * N); i++) {
		scanf("%d", &mat_h[i]);
	}

}

int fatorBranchingNivelDesejado(int nivelDesejado, int N){

	return N-nivelDesejado+1;
}

unsigned int calculaNPrefixos(int nivelPrefixo, int nVertice) {
	unsigned int x = nVertice - 1;
	int i;
	for (i = 1; i < nivelPrefixo-1; ++i) {
		x *= nVertice - i-1;
	}
	return x;
}

unsigned int calculaNPrefixosNivelDesejado(int nivelInicial,int nivelDesejado, int nVertice) {

	int nivelBusca = nivelInicial+1;
	int i;
	unsigned int nprefixos = 1;

	for (i = nivelBusca; i <=nivelDesejado; ++i) {
		nprefixos *= fatorBranchingNivelDesejado(i,N);
	}
	return nprefixos;
	
}


void fillFixedPaths(short* preFixo, const int nivelPrefixo) {
	char flag[16];
	int vertice[16]; //representa o ciclo
	int cont = 0;
	int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2


	for (i = 0; i < N; ++i) {
		flag[i] = 0;
		vertice[i] = -1;
	}

	vertice[0] = 0; //aqui!!!! vertice[nivel] = idx vflag[idx] = 1
	flag[0] = 1;
	nivel = 1;
	while (nivel >= 1) { // modificar aqui se quiser comecar a busca de determinado nivel

		if (vertice[nivel] != -1) {
			flag[vertice[nivel]] = 0;
		}

		do {
			vertice[nivel]++;
		} while (vertice[nivel] < N && flag[vertice[nivel]]); //


		if (vertice[nivel] < N) { //vertice[x] vertice no nivel x


			flag[vertice[nivel]] = 1;
			nivel++;

			if (nivel == nivelPrefixo) {
				for (i = 0; i < nivelPrefixo; ++i) {
					preFixo[cont * nivelPrefixo + i] = vertice[i];
//					printf("%d ", vertice[i]);
				}
//				printf("\n");
				cont++;
				nivel--;
			}
		} else {
			vertice[nivel] = -1;
			nivel--;
		}//else
	}//while
}

unsigned int finalDFS(const int N, const short *preFixos, const int idx, int *melhor_sol, const int nPreFixos, const int nivelPrefixo){
	
	int flag[16];
	int vertice[16]; //representa o ciclo
	
	int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
	int custo=0;
	unsigned int qtd_solucoes_thread = 0;
	int UB_local = INFINITO;
	int nivelGlobal = nivelPrefixo;

	if (idx < nPreFixos) { //(@)botar algo com vflag aqui, pois do jeito que esta algumas threads tentarao descer.
			
		for (i = 0; i < N; ++i) {
			vertice[i] = _VAZIO_;
			flag[i] = _NAO_VISITADO_;
		}
		
		vertice[0] = 0;
		flag[0] = _VISITADO_;
		custo= ZERO;
		
		for (i = 1; i < nivelGlobal; ++i) {
			vertice[i] = preFixos[idx * nivelGlobal + i];
			flag[vertice[i]] = _VISITADO_;
			custo += mat_h(vertice[i-1],vertice[i]);
		}
		
		nivel=nivelPrefixo;


		while (nivel >= nivelGlobal ) { // modificar aqui se quiser comecar a busca de determinado nivel

			if (vertice[nivel] != _VAZIO_) {
				flag[vertice[nivel]] = _NAO_VISITADO_;
				custo -= mat_h(vertice[anterior(nivel)],vertice[nivel]);
			}

			do {
				vertice[nivel]++;
			} while (vertice[nivel] < N && flag[vertice[nivel]]); //


			if (vertice[nivel] < N) { //vertice[x] vertice no nivel x
				custo += mat_h(vertice[anterior(nivel)],vertice[nivel]);
				flag[vertice[nivel]] = _VISITADO_;
				nivel++;

				if (nivel == N) { //se o vértice do nível for == N, entao formou o ciclo e vc soma peso + vertice anterior -> inicio
						
					++qtd_solucoes_thread;

					if (custo + mat_h(vertice[anterior(nivel)],0) < UB_local) {
						UB_local = custo + mat_h(vertice[anterior(nivel)],0);
					}
					nivel--;
				}
				//else {
					//if (custo > custoMin_d[0])
						//nivel--; //poda, LB maior que UB
				//}
			}
			else {
				vertice[nivel] = _VAZIO_;
				nivel--;
			}//else
		}//while

		// sols_d[idx] = qtd_solucoes_thread;
		// melhorSol_d[idx] = UB_local;

	}//dfs

	if(UB_local < (*melhor_sol)){
		*melhor_sol = UB_local;
	}
	return qtd_solucoes_thread;
}

int dfs2(const short *preFixos, const int idx, const int nivelInicial, const int nivelDesejado, int *otimo_global) {

	//pode tirar o idx

	register int flag[16];
	register int vertice[16]; //representa o ciclo
	register int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
	
	unsigned int qtd_solucoes_local = 0;
	unsigned int qtd_solucoes_filho = 0;

	int cont = 0;

	unsigned int qtd_prefixos_locais =  calculaNPrefixosNivelDesejado(nivelInicial,nivelDesejado,N);

	int melhor_sol = INFINITO;

	short *path_local;

	path_local = (short*)malloc(sizeof(short) * nivelDesejado *  qtd_prefixos_locais);
		

		for (i = 0; i < N; ++i) {
			vertice[i] = _VAZIO_;
			flag[i] = _NAO_VISITADO_;
		}
		
		vertice[0] = 0;
		flag[0] = _VISITADO_;
		custo= ZERO;
		
		for (i = 1; i < nivelInicial; ++i) {
			vertice[i] = preFixos[idx * nivelInicial + i];
			flag[vertice[i]] = _VISITADO_;
		}
		
		nivel=nivelInicial;

		while (nivel >= nivelInicial ) { // modificar aqui se quiser comecar a busca de determinado nivel

			if (vertice[nivel] != _VAZIO_) {
				flag[vertice[nivel]] = _NAO_VISITADO_;
			}

			do {
				vertice[nivel]++;
			} while (vertice[nivel] < N && flag[vertice[nivel]]); //


			if (vertice[nivel] < N) { //vertice[x] vertice no nivel x
			
				flag[vertice[nivel]] = _VISITADO_;
				nivel++;

				if (nivel == nivelDesejado) { //se o vértice do nível for == N, entao formou o ciclo e vc soma peso + vertice anterior -> inicio
						
					for (i = 0; i < nivelDesejado; ++i) {
						path_local[cont * nivelDesejado + i] = vertice[i];
					}
					++cont;
					nivel--;
				}
		
			}
			else {
				vertice[nivel] = _VAZIO_;
				nivel--;
			}//else
		}//while

	
	// for(int pref = 0; pref < qtd_prefixos_locais; ++pref){
	// 	for(int j = 0; j<nivelDesejado;++j){

	// 		printf(" %d ", path_local[pref*nivelDesejado + j]);
	// 	}
	// 	printf("\n");
	// }

	for(int pref = 0; pref < qtd_prefixos_locais; ++pref){
		qtd_solucoes_filho = finalDFS(N, path_local,pref, &melhor_sol, qtd_prefixos_locais,nivelDesejado);
		
		// printf("\nQtd de sols encontrada pelo prefixo %d: %d.\n",pref,qtd_solucoes_filho);

		qtd_solucoes_local+=qtd_solucoes_filho;
	
	}

	if(melhor_sol < (*otimo_global)){
		*otimo_global = melhor_sol;
	}

	free(path_local);

	return qtd_solucoes_local;

}//dfs2




int main() {

	read();

	
	int otimo_global = INFINITO;
	int qtd_sols_global = ZERO;

    
	int nivelPreFixos = 5;//Numero de niveis prefixados; o que nos permite utilizar mais threads. 
	int nivelDesejado = 8;

	unsigned int nPreFixos = calculaNPrefixos(nivelPreFixos,N);


	short * path_h = (short*) malloc(sizeof(short) * nPreFixos * nivelPreFixos);
	

	
	fillFixedPaths(path_h, nivelPreFixos);

	printf("\nNivel inicial: %d.", nivelPreFixos);
	printf("\nQuantidade de prefixos no nivel inicial: %d\n", nPreFixos);

	// for(int i = 0; i<nPreFixos;++i){
	// 	for(int j = 0; j<nivelPreFixos;++j )
	// 		printf(" %d ", path_h[i*nivelPreFixos+j]);
	// 	printf("\n");
	// }


	printf("\n Nivel desejado: %d.", nivelDesejado);
	printf("\n Prefixos individuais por pai no nivel desejado: %d .", calculaNPrefixosNivelDesejado(nivelPreFixos,nivelDesejado,N));
	printf("\n Prefixos totais no nivel desejado: %d .\n", calculaNPrefixosNivelDesejado(nivelPreFixos,nivelDesejado,N)*nPreFixos);


	//para nprefixos nivel inicial, imprimir as raizes de dfs descendentes criados
	for(int vertex = 0; vertex < nPreFixos; ++vertex){
		qtd_sols_global += dfs2(path_h, vertex, nivelPreFixos,nivelDesejado, &otimo_global) ;
	}

	printf("\n Quantidade de solucoes Global: %d.\n", qtd_sols_global);
	printf("\nOtimo global: %d. \n", otimo_global);
	


	return 0;
}
