
/*
 * main.c
 * SBLP 2014, special issue.
 * 
 *
 *  */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define mat_h(i,j) mat_h[i*N+j]
#define mat_d(i,j) mat_d[i*N_l+j]
#define mat_block(i,j) mat_block[i*N_l+j]
#define proximo(x) x+1
#define anterior(x) x-1
#define MAX 8192
#define INFINITO 999999
#define ZERO 0
#define ONE 1

#define _VAZIO_      -1
#define _VISITADO_    1
#define _NAO_VISITADO_ 0

int qtd = 0;
int custo = 0;
int N;
int melhor = INFINITO;
int upper_bound;

int mat_h[MAX];


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}



void read() {
	int i;
	//scanf("%d", &upper_bound);
	scanf("%d", &N);
	for (i = 0; i < (N * N); i++) {
		scanf("%d", &mat_h[i]);
	}

}

int fatorBranchingNivelDesejado(int nivelDesejado, int N){

	return N-nivelDesejado+1;
}

unsigned int calculaNPrefixos(int nivelPrefixo, int nVertice) {
	unsigned int x = nVertice - 1;
	int i;
	for (i = 1; i < nivelPrefixo-1; ++i) {
		x *= nVertice - i-1;
	}
	return x;
}

unsigned int calculaNPrefixosNivelDesejado(int nivelInicial,int nivelDesejado, int nVertice) {

	int nivelBusca = nivelInicial+1;
	int i;
	unsigned int nprefixos = 1;

	for (i = nivelBusca; i <=nivelDesejado; ++i) {
		nprefixos *= fatorBranchingNivelDesejado(i,N);
	}
	return nprefixos;
	
}


void fillFixedPaths(short* preFixo, const int nivelPrefixo) {
	char flag[16];
	int vertice[16]; //representa o ciclo
	int cont = 0;
	int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2


	for (i = 0; i < N; ++i) {
		flag[i] = 0;
		vertice[i] = -1;
	}

	vertice[0] = 0; //aqui!!!! vertice[nivel] = idx vflag[idx] = 1
	flag[0] = 1;
	nivel = 1;
	while (nivel >= 1) { // modificar aqui se quiser comecar a busca de determinado nivel

		if (vertice[nivel] != -1) {
			flag[vertice[nivel]] = 0;
		}

		do {
			vertice[nivel]++;
		} while (vertice[nivel] < N && flag[vertice[nivel]]); //


		if (vertice[nivel] < N) { //vertice[x] vertice no nivel x


			flag[vertice[nivel]] = 1;
			nivel++;

			if (nivel == nivelPrefixo) {
				for (i = 0; i < nivelPrefixo; ++i) {
					preFixo[cont * nivelPrefixo + i] = vertice[i];
//					printf("%d ", vertice[i]);
				}
//				printf("\n");
				cont++;
				nivel--;
			}
		} else {
			vertice[nivel] = -1;
			nivel--;
		}//else
	}//while
}

short * gerar_prefixos_iniciais(int nivelPreFixos, int nivelDesejado, int nPreFixos){
    	short * path_h = (short*) malloc(sizeof(short) * nPreFixos * nivelPreFixos);
	fillFixedPaths(path_h, nivelPreFixos);
	return path_h;
}

__global__ void dfs_final(int N, int *mat_d, short *preFixos_d, int nPrefixosNivelDesejado, int nivelDesejado, unsigned  int *sols_d,  int *melhorSol_d, int salto) {

	register int idx = blockIdx.x * blockDim.x + threadIdx.x;
	register int flag[16];
	register int vertice[16]; //representa o ciclo
	
	register int N_l = N;
	
	register int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
	register int custo;
	register int qtd_solucoes_thread = 0;
	register int UB_local = INFINITO;
	register int nivelGlobal = nivelDesejado;

	//int idxGlobal = idBlocoPai*blockDim.x+idx;

	int idxGlobal = salto+idx;

	if (idxGlobal < nPrefixosNivelDesejado) {
			
		for (i = 0; i < N_l; ++i) {
			vertice[i] = _VAZIO_;
			flag[i] = _NAO_VISITADO_;
		}
		
		vertice[0] = 0;
		flag[0] = _VISITADO_;
		custo= ZERO;
		
		for (i = 1; i < nivelGlobal; ++i) {
			vertice[i] = preFixos_d[idx * nivelGlobal + i];
			flag[vertice[i]] = _VISITADO_;
			custo += mat_d(vertice[i-1],vertice[i]);
		}
		
		nivel=nivelGlobal;

	

		while (nivel >= nivelGlobal ) { // modificar aqui se quiser comecar a busca de determinado nivel

			if (vertice[nivel] != _VAZIO_) {
				flag[vertice[nivel]] = _NAO_VISITADO_;
				custo -= mat_d(vertice[anterior(nivel)],vertice[nivel]);
			}

			do {
				vertice[nivel]++;
			} while (vertice[nivel] < N_l && flag[vertice[nivel]]); //


			if (vertice[nivel] < N_l) { //vertice[x] vertice no nivel x
				custo += mat_d(vertice[anterior(nivel)],vertice[nivel]);
				flag[vertice[nivel]] = _VISITADO_;
				nivel++;

				if (nivel == N_l) { //se o vértice do nível for == N, entao formou o ciclo e vc soma peso + vertice anterior -> inicio
						
					++qtd_solucoes_thread;

					if (custo + mat_d(vertice[anterior(nivel)],0) < UB_local) {
						UB_local = custo + mat_d(vertice[anterior(nivel)],0);
					}
					nivel--;
				}
				//else {
					//if (custo > custoMin_d[0])
						//nivel--; //poda, LB maior que UB
				//}
			}
			else {
				vertice[nivel] = _VAZIO_;
				nivel--;
			}//else
		}//while

		sols_d[idx] = qtd_solucoes_thread;
		melhorSol_d[idx] = UB_local;

	}//dfs

}//kernel




__global__ void dfs_intermediario_cuda(int N, int *mat_d, short *preFixos_d,short *preFixos_novos_d,unsigned int nPreFixos,unsigned int qtd_prefixos_segundo_dfs,unsigned int qtd_prefixos_locais,int nivelInicial, int nivelDesejado, unsigned int *qtd_sols_d, int *melhor_sol_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int flag[16];
	int vertice[16]; //representa o ciclo
	
	int N_l = N;
	
	int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
	int cont = 0;
	
	nivel=nivelInicial;

	if (idx < nPreFixos) { //(@)botar algo com vflag aqui, pois do jeito que esta algumas threads tentarao descer.
			
	
		for (i = 0; i < N_l; ++i) {
			vertice[i] = _VAZIO_;
			flag[i] = _NAO_VISITADO_;
		}
		
		vertice[0] = 0;
		flag[0] = _VISITADO_;
		

		for (i = 1; i < nivel; ++i) {
			vertice[i] = preFixos_d[idx * nivelInicial + i];
			flag[vertice[i]] = _VISITADO_;
		}
		

		// for (i = 0; i < N; ++i) {
		// 	vertice[i] = _VAZIO_;
		// 	flag[i] = _NAO_VISITADO_;
		// }
		
		// vertice[0] = 0;
		// flag[0] = _VISITADO_;
	
		
		// for (i = 1; i < nivelInicial; ++i) {
		// 	vertice[i] = preFixos[idx * nivelInicial + i];
		// 	flag[vertice[i]] = _VISITADO_;
		// }
		
	

		while (nivel >= nivelInicial) { // modificar aqui se quiser comecar a busca de determinado nivel

			if (vertice[nivel] != _VAZIO_) {
				flag[vertice[nivel]] = _NAO_VISITADO_;
			}

			do {
				vertice[nivel]++;
			} while (vertice[nivel] < N_l && flag[vertice[nivel]]); //


			if (vertice[nivel] < N_l) { //vertice[x] vertice no nivel x
				flag[vertice[nivel]] = _VISITADO_;
				nivel++;

				if (nivel == nivelDesejado) { //se o vértice do nível for == N, entao formou o ciclo e vc soma peso + vertice anterior -> inicio
						
					
					for (i = 0; i < nivelDesejado; ++i) {

						preFixos_novos_d[(idx*qtd_prefixos_locais*nivelDesejado) + (cont*nivelDesejado)+i] = vertice[i];
					}
					
					++cont;

					nivel--;
				}
				//else {
					//if (custo > custoMin_d[0])
						//nivel--; //poda, LB maior que UB
				//}
			}
			else {
				vertice[nivel] = _VAZIO_;
				nivel--;
			}//else
		}//while

	}//dfs
	
	__syncthreads();

	if(threadIdx.x == 0){
		

		//int qtd_prefixos_nivel_desejado = qtd_prefixos_locais * nPreFixos;
	    int block_size = 192;
		int n_blocks = (blockDim.x*qtd_prefixos_locais) / block_size + (blockDim.x % block_size == 0 ? 0 : 1);
//printf("\nSou a thread mestra do bloco %d criando %d blocks", blockIdx.x, n_blocks);
		// int saltoPrefixos = qtd_prefixos_locais*nivelDesejado*blockIdx.x*blockDim.x;
		// int saltoSolucoes = qtd_prefixos_locais * blockIdx.x*blockDim.x;
		// int saltoMelhorSol = qtd_prefixos_locais *blockIdx.x*blockDim.x; 
		
		 int saltoPrefixos = n_blocks*block_size*nivelDesejado*blockIdx.x;
		 int saltoSolucoes = n_blocks*block_size*blockIdx.x;
		 int saltoMelhorSol = n_blocks*block_size*blockIdx.x; 
		 int salto = n_blocks*block_size*blockIdx.x;

	    dfs_final<<<n_blocks,block_size>>>(N_l, mat_d, (preFixos_novos_d+saltoPrefixos), qtd_prefixos_segundo_dfs,  nivelDesejado , (qtd_sols_d+saltoSolucoes),(melhor_sol_d+saltoMelhorSol),salto);
//l__ void dfs_final(int N, int *mat_d, short *preFixos_d, int nPrefixosNivelDesejado, int nivelDesejado, unsigned  int *sols_d,  int *melhorSol_d) {

	
	}

}//kernel



/*
	Irei aqui alocar a matriz de custos, os prefixos do primeiro DFS e memoria suficiente pros prefixos do segundo DFS.
		NAO IREI FAZER MALLOCS EM TEMPO DE EXECUCAO, pos nao sei se o fusion possui isso
*/
void call_cuda_DFSIntermediario(short *path_h, unsigned int nPreFixos, unsigned int qtd_prefixos_segundo_dfs, int nivelPreFixos,int nivelDesejado){

	short *path_d;
	short *path_second_dfs_d;
	short *path_second_dfs_h = (short*)malloc(sizeof(short)*qtd_prefixos_segundo_dfs*nivelDesejado);

	unsigned int qtd_prefixos_locais = calculaNPrefixosNivelDesejado(nivelPreFixos,nivelDesejado,N);
	unsigned int qtd_sols_global = 0;

	int *mat_d;

	unsigned int *qtd_sols_h = (unsigned int*)malloc(sizeof(unsigned int)*qtd_prefixos_segundo_dfs);
	unsigned int *qtd_sols_d;

	int *melhor_sol_h = (int*)malloc(sizeof(int)*qtd_prefixos_segundo_dfs);
	int *melhor_sol_d;

	int block_size =192; //number threads in a block
	int n_blocks = nPreFixos / block_size + (nPreFixos % block_size == 0 ? 0 : 1); // # of blocks


	int otimo_global = INFINITO;
//	for(int i = 0; i<nPreFixos;++i){
//		for(int j = 0; j<nivelPreFixos;++j )
//			printf(" %d ", path_h[i*nivelPreFixos+j]);
//		printf("\n");
//	}

      	printf("\nQuantidade de prefixos por raiz: %d\n", qtd_prefixos_locais);


       	printf("\nQuantidade de prefixos no nivel desejado: %d\n", qtd_prefixos_segundo_dfs);


	HANDLE_ERROR( hipMalloc((void **) &mat_d, N * N * sizeof(int)));
	HANDLE_ERROR( hipMalloc((void **) &path_d, nPreFixos*nivelPreFixos*sizeof(short)));
	HANDLE_ERROR( hipMalloc((void **) &path_second_dfs_d, qtd_prefixos_segundo_dfs*nivelDesejado*sizeof(short)));

	HANDLE_ERROR( hipMalloc((void **) &qtd_sols_d, sizeof(int)*qtd_prefixos_segundo_dfs));
	HANDLE_ERROR( hipMalloc((void **) &melhor_sol_d, sizeof(int)*qtd_prefixos_segundo_dfs));

	HANDLE_ERROR( hipMemcpy(mat_d, mat_h, N * N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipMemcpy(path_d, path_h, nPreFixos*nivelPreFixos*sizeof(short), hipMemcpyHostToDevice));

	
	hipDeviceSynchronize();
	dfs_intermediario_cuda<<< n_blocks,block_size >>>(N,mat_d,path_d, path_second_dfs_d, nPreFixos , qtd_prefixos_segundo_dfs,qtd_prefixos_locais,nivelPreFixos, nivelDesejado,qtd_sols_d,melhor_sol_d);

	hipDeviceSynchronize();
	HANDLE_ERROR( hipMemcpy(path_second_dfs_h, path_second_dfs_d, qtd_prefixos_segundo_dfs*nivelDesejado*sizeof(short), hipMemcpyDeviceToHost));

        HANDLE_ERROR( hipMemcpy(qtd_sols_h,qtd_sols_d, sizeof(unsigned int)*qtd_prefixos_segundo_dfs, hipMemcpyDeviceToHost));
       HANDLE_ERROR( hipMemcpy(melhor_sol_h, melhor_sol_d, sizeof(int)*qtd_prefixos_segundo_dfs, hipMemcpyDeviceToHost));
	
	 for(int i = 0; i<qtd_prefixos_segundo_dfs; ++i){
	 	qtd_sols_global+=qtd_sols_h[i];
	 	if(melhor_sol_h[i]<otimo_global)
	 		otimo_global = melhor_sol_h[i];
	 	//printf("\nSolucoes encontradas pela thread %d: %d", i, sols_h[i]);	
	 	//printf("\n\tMelhor solucao encontrada pela thread %d: %d", i, melhorSol_h[i]);
	 }

	printf("\nQuantidade de solucoes global: %d. \n Otimo global: %d.\n", qtd_sols_global, otimo_global);

//	for(int i = 0; i<qtd_prefixos_segundo_dfs;++i){
//	 	for(int j = 0; j<nivelDesejado;++j )
//	 		printf(" %d ", path_second_dfs_h[i*nivelDesejado+j]);
//	 	printf("\n");
//	 }

}


int main() {

	read();

	int nivelPreFixos = 5;//Numero de niveis prefixados; o que nos permite utilizar mais threads. 
	int nivelDesejado = 7;

	short *path_h;

	//unsigned int qtd_sols_global = 0;

	//int otimo_global = INFINITO;

	unsigned int nPreFixos = calculaNPrefixos(nivelPreFixos,N);

	unsigned int qtd_prefixos_segundo_dfs;

	printf("\nNivel inicial: %d.", nivelPreFixos);
	printf("\nQuantidade de prefixos no nivel inicial: %d\n", nPreFixos);

	path_h = gerar_prefixos_iniciais(nivelPreFixos,nivelDesejado,nPreFixos);

	qtd_prefixos_segundo_dfs = nPreFixos * calculaNPrefixosNivelDesejado(nivelPreFixos,nivelDesejado,N);

	call_cuda_DFSIntermediario(path_h, nPreFixos, qtd_prefixos_segundo_dfs, nivelPreFixos, nivelDesejado);



	exit(1);

	return 0;
}
