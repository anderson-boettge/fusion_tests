#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//I need to check how to do this in the compiler
#define mat_h(i,j) mat_h[i*N+j]
#define mat_d(i,j) mat_d[i*N_l+j]
#define mat_block(i,j) mat_block[i*N_l+j]
#define proximo(x) x+1
#define anterior(x) x-1
#define MAX 8192
#define INFINITO 999999
#define ZERO 0
#define ONE 1

#define _VAZIO_      -1
#define _VISITADO_    1
#define _NAO_VISITADO_ 0

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}
                           
#ifdef __cplusplus
extern "C"
{
#endif
/**global variables accelerator obj**/
int nivelPreFixos, nivelDesejado;
int N;
  
__global__ void dfs_final(int N, int *mat_d, int *preFixos_d, int nPrefixosNivelDesejado, int nivelDesejado, int *sols_d,  int *melhorSol_d, int salto) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int flag[16];
	int vertice[16]; //representa o ciclo
	
	int N_l = N;
	
	int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
	int custo;
	int qtd_solucoes_thread = 0;
	int UB_local = INFINITO;
	int nivelGlobal = nivelDesejado;

	int idxGlobal = salto+idx;


	if (idxGlobal < nPrefixosNivelDesejado) {
		for (i = 0; i < N_l; ++i) {
			vertice[i] = _VAZIO_;
			flag[i] = _NAO_VISITADO_;
		}
		
		vertice[0] = 0;
		flag[0] = _VISITADO_;
		custo= ZERO;
		
		for (i = 1; i < nivelGlobal; ++i) {
			vertice[i] = preFixos_d[idx * nivelGlobal + i];
			flag[vertice[i]] = _VISITADO_;
			custo += mat_d(vertice[i-1],vertice[i]);
		}
		
		nivel=nivelGlobal;

		while (nivel >= nivelGlobal ) { // modificar aqui se quiser comecar a busca de determinado nivel

			if (vertice[nivel] != _VAZIO_) {
				flag[vertice[nivel]] = _NAO_VISITADO_;
				custo -= mat_d(vertice[anterior(nivel)],vertice[nivel]);
			}

			do {
				vertice[nivel]++;
			} while (vertice[nivel] < N_l && flag[vertice[nivel]]); //


			if (vertice[nivel] < N_l) { //vertice[x] vertice no nivel x
				custo += mat_d(vertice[anterior(nivel)],vertice[nivel]);
				flag[vertice[nivel]] = _VISITADO_;
				nivel++;

				if (nivel == N_l) { //se o vértice do nível for == N, entao formou o ciclo e vc soma peso + vertice anterior -> inicio
						
					++qtd_solucoes_thread;

					if (custo + mat_d(vertice[anterior(nivel)],0) < UB_local) {
						UB_local = custo + mat_d(vertice[anterior(nivel)],0);
					}
					nivel--;
				}
			}
			else {
				vertice[nivel] = _VAZIO_;
				nivel--;
			}//else
		}//while

		sols_d[idx] = qtd_solucoes_thread;
		melhorSol_d[idx] = UB_local;

	}//dfs

}//kernel


__global__ void dfs_intermediario_cuda(int N, int *mat_d, int *preFixos_d, int *preFixos_novos_d, int nPreFixos, int qtd_prefixos_segundo_dfs, int qtd_prefixos_locais,int nivelInicial, int nivelDesejado, int *qtd_sols_d, int *melhor_sol_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int flag[16];
	int vertice[16]; //representa o ciclo
	
	int N_l = N;
	
	int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
	int cont = 0;
	
	nivel=nivelInicial;

	if (idx < nPreFixos) { //(@)botar algo com vflag aqui, pois do jeito que esta algumas threads tentarao descer.
	
		for (i = 0; i < N_l; ++i) {
			vertice[i] = _VAZIO_;
			flag[i] = _NAO_VISITADO_;
		}
		
		vertice[0] = 0;
		flag[0] = _VISITADO_;

		for (i = 1; i < nivel; ++i) {
			vertice[i] = preFixos_d[idx * nivelInicial + i];
			flag[vertice[i]] = _VISITADO_;
		}

		while (nivel >= nivelInicial) { // modificar aqui se quiser comecar a busca de determinado nivel

			if (vertice[nivel] != _VAZIO_) {
				flag[vertice[nivel]] = _NAO_VISITADO_;
			}

			do {
				vertice[nivel]++;
			} while (vertice[nivel] < N_l && flag[vertice[nivel]]); //

			if (vertice[nivel] < N_l) { //vertice[x] vertice no nivel x
				flag[vertice[nivel]] = _VISITADO_;
				nivel++;
				if (nivel == nivelDesejado) { //se o vértice do nível for == N, entao formou o ciclo e vc soma peso + vertice anterior -> inicio
					for (i = 0; i < nivelDesejado; ++i) {
						preFixos_novos_d[(idx*qtd_prefixos_locais*nivelDesejado) + (cont*nivelDesejado)+i] = vertice[i];
					}
					++cont;
					nivel--;
				}
			}
			else {
				vertice[nivel] = _VAZIO_;
				nivel--;
			}//else
		}//while

	}//dfs
	
	__syncthreads();

	if(threadIdx.x == 0){
	    int block_size = 192;
	    int n_blocks = (blockDim.x*qtd_prefixos_locais) / block_size + (blockDim.x % block_size == 0 ? 0 : 1);
            int saltoPrefixos = n_blocks*block_size*nivelDesejado*blockIdx.x;
	    int saltoSolucoes = n_blocks*block_size*blockIdx.x;
	    int saltoMelhorSol = n_blocks*block_size*blockIdx.x; 
	    int salto = n_blocks*block_size*blockIdx.x;

	    dfs_final<<<n_blocks,block_size>>>(N_l, mat_d, (preFixos_novos_d+saltoPrefixos), qtd_prefixos_segundo_dfs,  nivelDesejado , (qtd_sols_d+saltoSolucoes),(melhor_sol_d+saltoMelhorSol),salto);
	}
}//kernel


//local variables unit DFS_intermediario
int nPreFixos;
int qtd_prefixos_segundo_dfs;
int qtd_prefixos_locais;

int *path_second_dfs_d;
int *qtd_sols_d;
int *melhor_sol_d;
int *path_d;
int *mat_d;


void setData_c(int Nh, int* mat_h, int nivelP, int nivelD, int* path_h, int nPreFix, int qtd_segundo_dfs, int qtd_locais, int* qtd_sols_h, int* melhor_sol_h){
printf("SETDATA entrou");
    N = Nh;
    nPreFixos = nPreFix;
    qtd_prefixos_segundo_dfs = qtd_segundo_dfs;
    nivelPreFixos = nivelP;
    nivelDesejado = nivelD;
    qtd_prefixos_locais = qtd_locais;
    
    HANDLE_ERROR( hipMalloc((void **) &mat_d, N * N * sizeof(int)));
    HANDLE_ERROR( hipMalloc((void **) &path_d, nPreFixos*nivelPreFixos*sizeof(int)));
    HANDLE_ERROR( hipMalloc((void **) &path_second_dfs_d, qtd_prefixos_segundo_dfs*nivelDesejado*sizeof(int)));

    HANDLE_ERROR( hipMalloc((void **) &qtd_sols_d, sizeof(int)*qtd_prefixos_segundo_dfs));
    HANDLE_ERROR( hipMalloc((void **) &melhor_sol_d, sizeof(int)*qtd_prefixos_segundo_dfs));

    HANDLE_ERROR( hipMemcpy(mat_d, mat_h, N * N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR( hipMemcpy(path_d, path_h, nPreFixos*nivelPreFixos*sizeof(int), hipMemcpyHostToDevice));
    
    printf("SETDATA OK");
}

void k_dfs_intermediario(int n_blocks, int block_size){
  printf("SETDATA KERNEL ENTROU");
    hipDeviceSynchronize();
    dfs_intermediario_cuda<<< n_blocks,block_size >>>(N,mat_d,path_d, path_second_dfs_d, nPreFixos , qtd_prefixos_segundo_dfs,qtd_prefixos_locais,nivelPreFixos, nivelDesejado,qtd_sols_d,melhor_sol_d);
}

int* getResult(int* result){
  int qtd_sols_global, otimo_global;
  int *qtd_sols_h = (int*)malloc(sizeof(int)*qtd_prefixos_segundo_dfs);
  int *melhor_sol_h = (int*)malloc(sizeof(int)*qtd_prefixos_segundo_dfs);
  
  hipDeviceSynchronize();
  
  HANDLE_ERROR( hipMemcpy(qtd_sols_h,qtd_sols_d, sizeof(int)*qtd_prefixos_segundo_dfs, hipMemcpyDeviceToHost));
  HANDLE_ERROR( hipMemcpy(melhor_sol_h, melhor_sol_d, sizeof(int)*qtd_prefixos_segundo_dfs, hipMemcpyDeviceToHost));
	
  for(int i = 0; i<qtd_prefixos_segundo_dfs; ++i){
      qtd_sols_global+=qtd_sols_h[i];
      if(melhor_sol_h[i]<otimo_global)
	otimo_global = melhor_sol_h[i];
  }
  hipFree(mat_d);
  hipFree(path_d);
  hipFree(path_second_dfs_d);
  hipFree(qtd_sols_d);
  hipFree(melhor_sol_d);
  
  result[0]=qtd_sols_global;
  result[1]=otimo_global;
  return result;
}

#ifdef __cplusplus
}
#endif

