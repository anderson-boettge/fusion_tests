#include<hip/hip_runtime.h>


#define mat_h(i,j) mat_h[i*N+j]
#define mat_d(i,j) mat_d[i*N_l+j]
#define mat_block(i,j) mat_block[i*N_l+j]
#define proximo(x) x+1
#define anterior(x) x-1
#define MAX 8192
#define INFINITO 999999
#define ZERO 0
#define ONE 1

#define _VAZIO_      -1
#define _VISITADO_    1
#define _NAO_VISITADO_ 0


__global__ void dfs_final(int N, int *mat_d, int *preFixos_d, int nPrefixosNivelDesejado, int nivelDesejado, int *sols_d,  int *melhorSol_d, int salto) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int flag[16];
	int vertice[16]; //representa o ciclo
	
	int N_l = N;
	
	int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
	int custo;
	int qtd_solucoes_thread = 0;
	int UB_local = INFINITO;
	int nivelGlobal = nivelDesejado;

	int idxGlobal = salto+idx;


	if (idxGlobal < nPrefixosNivelDesejado) {
		for (i = 0; i < N_l; ++i) {
			vertice[i] = _VAZIO_;
			flag[i] = _NAO_VISITADO_;
		}
		
		vertice[0] = 0;
		flag[0] = _VISITADO_;
		custo= ZERO;
		
		for (i = 1; i < nivelGlobal; ++i) {
			vertice[i] = preFixos_d[idx * nivelGlobal + i];
			flag[vertice[i]] = _VISITADO_;
			custo += mat_d(vertice[i-1],vertice[i]);
		}
		
		nivel=nivelGlobal;

		while (nivel >= nivelGlobal ) { // modificar aqui se quiser comecar a busca de determinado nivel

			if (vertice[nivel] != _VAZIO_) {
				flag[vertice[nivel]] = _NAO_VISITADO_;
				custo -= mat_d(vertice[anterior(nivel)],vertice[nivel]);
			}

			do {
				vertice[nivel]++;
			} while (vertice[nivel] < N_l && flag[vertice[nivel]]); //


			if (vertice[nivel] < N_l) { //vertice[x] vertice no nivel x
				custo += mat_d(vertice[anterior(nivel)],vertice[nivel]);
				flag[vertice[nivel]] = _VISITADO_;
				nivel++;

				if (nivel == N_l) { //se o vértice do nível for == N, entao formou o ciclo e vc soma peso + vertice anterior -> inicio
						
					++qtd_solucoes_thread;

					if (custo + mat_d(vertice[anterior(nivel)],0) < UB_local) {
						UB_local = custo + mat_d(vertice[anterior(nivel)],0);
					}
					nivel--;
				}
			}
			else {
				vertice[nivel] = _VAZIO_;
				nivel--;
			}//else
		}//while

		sols_d[idx] = qtd_solucoes_thread;
		melhorSol_d[idx] = UB_local;

	}//dfs

}//kernel

__device__ void call_dfs_final(int n_blocks, int block_size, int N, int *mat_d, int *preFixos_d, int nPrefixosNivelDesejado, int nivelDesejado, int *sols_d,  int *melhorSol_d, int salto){
  
  
  dfs_final<<<n_blocks, block_size>>>(N, mat_d, preFixos_d, nPrefixosNivelDesejado,  nivelDesejado , sols_d,melhorSol_d,salto);
}